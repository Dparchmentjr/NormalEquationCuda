
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 2

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x;
           int j = threadIdx.y;

           C[i][j] = A[i][j] + B[i][j];
       }

__global__ void MatMult(double A[][N], double B[][N], double C[][N], int width){
               int i = threadIdx.x;
               int j = threadIdx.y;
               int k;
               double Pvalue = 0;

               for(k = 0; k < width; k++){
                   Pvalue += A[i][k] * B[k][j];
               }

               C[i][j] = Pvalue;
}

__global__ void MatTrans(double A[][N], double C[][N], int width, int height){
               int i = threadIdx.x;
               int j = threadIdx.y;

               C[i][j] = A[j][i];

}
int main(){

double A[N][N] = {{1,2},{3,4}};
double B[N][N] = {{1,2},{3,4}};
double C[N][N] = {{0,0},{0,0}};

double (*pA)[N], (*pB)[N], (*pC)[N];

hipMalloc((void**)&pA, (N*N)*sizeof(double));
hipMalloc((void**)&pB, (N*N)*sizeof(double));
hipMalloc((void**)&pC, (N*N)*sizeof(double));

hipMemcpy(pA, A, (N*N)*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(pB, B, (N*N)*sizeof(double), hipMemcpyHostToDevice);
hipMemcpy(pC, C, (N*N)*sizeof(double), hipMemcpyHostToDevice);

int numBlocks = 1;
dim3 threadsPerBlock(N,N);
//MatMult<<<numBlocks,threadsPerBlock>>>(pA, pB, pC,2);
MatTrans<<<numBlocks,threadsPerBlock>>>(pA, pC, 2, 2);

hipMemcpy(C, pC, (N*N)*sizeof(double), hipMemcpyDeviceToHost);

int i, j; printf("C = \n");
for(i=0;i<N;i++){
    for(j=0;j<N;j++){
        printf("%f ", C[i][j]);
    }
    printf("\n");
}

hipFree(pA);
hipFree(pB);
hipFree(pC);

printf("\n");
return 0;
}
